#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <vector>
#include <algorithm>
#include <map>
#include <cmath>

// #define NUM_MATRICES 31*80*80
#define NUM_MATRICES (927000)
#define MATRIX_SIZE 3
#define BLOCK_SIZE 4096

#define IS_PRINT_RUNTIME false
#define IS_PRINT_ERROR false

// 生成随机对称矩阵
void generateRandomSymmetricMatrices(float* matrices, int numMatrices) {
    for (int i = 0; i < numMatrices; ++i) {
        float* m = matrices + i * 9;
        for (int j = 0; j < 9; ++j) {
            m[j] = (float)rand() / RAND_MAX * 2.0f - 1.0f;
        }
        // Symmetrize matrix
        for (int row = 0; row < 3; ++row) {
            for (int col = row + 1; col < 3; ++col) {
                float avg = (m[row*3 + col] + m[col*3 + row]) / 2.0f;
                m[row*3 + col] = avg;
                m[col*3 + row] = avg;
            }
        }
    }
}

// 交换特征值和特征向量
__device__ void swap(float* a, float* b) {
    float temp = *a;
    *a = *b;
    *b = temp;
}

// 3x3对称矩阵的Jacobi特征分解
__device__ void jacobi3x3(const float* A, float* eigVals, float* eigVecs) {
    float V[9] = {1.0f, 0.0f, 0.0f,
                  0.0f, 1.0f, 0.0f,
                  0.0f, 0.0f, 1.0f};
    float B[9];
    for (int i = 0; i < 9; ++i) B[i] = A[i];

    const int maxIter = 50;
    const float epsilon = 1e-6f;

    for (int iter = 0; iter < maxIter; ++iter) {
        // Find max off-diagonal element
        int p = 0, q = 1;
        float maxVal = fabsf(B[1]);
        if (fabsf(B[2]) > maxVal) { p = 0; q = 2; maxVal = fabsf(B[2]); }
        if (fabsf(B[5]) > maxVal) { p = 1; q = 2; maxVal = fabsf(B[5]); }

        if (maxVal < epsilon) break;

        // Compute rotation angle
        float theta = 0.5f * atan2f(2 * B[p*3 + q], B[q*3 + q] - B[p*3 + p]);
        float c = cosf(theta);
        float s = sinf(theta);

        // Apply rotation to B
        float Bpp = B[p*3 + p];
        float Bqq = B[q*3 + q];
        float Bpq = B[p*3 + q];

        B[p*3 + p] = c*c*Bpp + s*s*Bqq - 2*c*s*Bpq;
        B[q*3 + q] = s*s*Bpp + c*c*Bqq + 2*c*s*Bpq;
        B[p*3 + q] = 0.0f;
        B[q*3 + p] = 0.0f;

        for (int r = 0; r < 3; ++r) {
            if (r != p && r != q) {
                float Brp = B[r*3 + p];
                float Brq = B[r*3 + q];
                B[r*3 + p] = c*Brp - s*Brq;
                B[p*3 + r] = B[r*3 + p];
                B[r*3 + q] = s*Brp + c*Brq;
                B[q*3 + r] = B[r*3 + q];
            }
        }

        // Update eigenvectors
        for (int r = 0; r < 3; ++r) {
            float Vrp = V[r*3 + p];
            float Vrq = V[r*3 + q];
            V[r*3 + p] = c*Vrp - s*Vrq;
            V[r*3 + q] = s*Vrp + c*Vrq;
        }
    }

    // Extract eigenvalues
    eigVals[0] = B[0];
    eigVals[1] = B[4];
    eigVals[2] = B[8];

    // Sort eigenvalues and eigenvectors
    if (eigVals[0] < eigVals[1]) {
        swap(&eigVals[0], &eigVals[1]);
        for (int r = 0; r < 3; ++r) swap(&V[r*3 + 0], &V[r*3 + 1]);
    }
    if (eigVals[0] < eigVals[2]) {
        swap(&eigVals[0], &eigVals[2]);
        for (int r = 0; r < 3; ++r) swap(&V[r*3 + 0], &V[r*3 + 2]);
    }
    if (eigVals[1] < eigVals[2]) {
        swap(&eigVals[1], &eigVals[2]);
        for (int r = 0; r < 3; ++r) swap(&V[r*3 + 1], &V[r*3 + 2]);
    }

    // Store eigenvectors
    for (int i = 0; i < 9; ++i) eigVecs[i] = V[i];
}

__global__ void eigenDecompositionKernel(const float* matrices, float* eigenvalues, float* eigenvectors, int numMatrices) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numMatrices) return;

    float matrix[9];
    for (int i = 0; i < 9; ++i) matrix[i] = matrices[idx*9 + i];

    float eigVals[3];
    float eigVecs[9];
    jacobi3x3(matrix, eigVals, eigVecs);

    for (int i = 0; i < 3; ++i) eigenvalues[idx*3 + i] = eigVals[i];
    for (int i = 0; i < 9; ++i) eigenvectors[idx*9 + i] = eigVecs[i];
}



std::tuple<double, double> testOnce(){
    srand(42); // 固定随机种子便于验证

    // 分配主机内存
    float* h_matrices = (float*)malloc(NUM_MATRICES * 9 * sizeof(float));
    float* h_eigenvalues = (float*)malloc(NUM_MATRICES * 3 * sizeof(float));
    float* h_eigenvectors = (float*)malloc(NUM_MATRICES * 9 * sizeof(float));

    // 生成随机对称矩阵
    generateRandomSymmetricMatrices(h_matrices, NUM_MATRICES);


    // 创建 CUDA 事件
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 记录起始时间
    hipEventRecord(start);

    auto start_time_cpu = std::chrono::high_resolution_clock::now();

    // 分配设备内存
    float *d_matrices, *d_eigenvalues, *d_eigenvectors;
    hipMalloc(&d_matrices, NUM_MATRICES * 9 * sizeof(float));
    hipMalloc(&d_eigenvalues, NUM_MATRICES * 3 * sizeof(float));
    hipMalloc(&d_eigenvectors, NUM_MATRICES * 9 * sizeof(float));

    // 拷贝数据到设备
    hipMemcpy(d_matrices, h_matrices, NUM_MATRICES * 9 * sizeof(float), hipMemcpyHostToDevice);

    // 启动核函数
    int blockSize = BLOCK_SIZE;
    int gridSize = (NUM_MATRICES + blockSize - 1) / blockSize;
    eigenDecompositionKernel<<<gridSize, blockSize>>>(d_matrices, d_eigenvalues, d_eigenvectors, NUM_MATRICES);

    // 拷贝回结果
    hipMemcpy(h_eigenvalues, d_eigenvalues, NUM_MATRICES * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_eigenvectors, d_eigenvectors, NUM_MATRICES * 9 * sizeof(float), hipMemcpyDeviceToHost);

    auto end_time_cpu = std::chrono::high_resolution_clock::now();
    auto duration_cpu = std::chrono::duration_cast<std::chrono::microseconds>(end_time_cpu - start_time_cpu);
    if(IS_PRINT_RUNTIME){
        printf("CPU 计算耗时: %.3f 毫秒\n", duration_cpu.count()/1000.0f);

    }
    

    // 记录结束时间
    hipEventRecord(stop);

    // 等待核函数完成
    hipEventSynchronize(stop);

    // 计算时间差
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    if(IS_PRINT_RUNTIME){
        printf("CUDA 计算耗时: %.3f 毫秒\n", milliseconds);
    }


    // 验证结果
    const int numTests = NUM_MATRICES;
    bool is_print_matrix = false;
    float all_error = 0.0f;
    for (int t = 0; t < numTests; ++t) {
        int idx = rand() % NUM_MATRICES;

        // 原始矩阵
        float* m = h_matrices + idx*9;


        // 特征值和特征向量
        float* vals = h_eigenvalues + idx*3;
        float* vecs = h_eigenvectors + idx*9;


        // 重构矩阵
        float reconstructed[9] = {0};
        for (int k = 0; k < 3; ++k) {
            float lambda = vals[k];
            for (int i = 0; i < 3; ++i) {
                for (int j = 0; j < 3; ++j) {
                    reconstructed[i*3 + j] += lambda * vecs[i*3 + k] * vecs[j*3 + k];
                }
            }
        }

        // 计算差异
        float diff = 0.0f;
        const float epsilon = 1e15f;
        for (int i = 0; i < 9; ++i) 
            diff += fabsf(reconstructed[i] - m[i]);
        auto diff_fourdot = (int)(diff * epsilon)/epsilon;
        all_error += diff_fourdot;

        if(is_print_matrix){
            printf("\n验证矩阵 %d:\n", idx);

            // print 原始矩阵
            printf("原始矩阵:\n");
            for (int i = 0; i < 3; ++i) {
                printf("[%8.4f %8.4f %8.4f]\n", m[i*3], m[i*3+1], m[i*3+2]);
            }
            
            // print 特征值和特征向量
            printf("\n特征值: %8.4f %8.4f %8.4f\n", vals[0], vals[1], vals[2]);
            printf("特征向量:\n");
            for (int i = 0; i < 3; ++i) {
                printf("[%8.4f %8.4f %8.4f]\n", vecs[i*3], vecs[i*3+1], vecs[i*3+2]);
            }

            // print 重构矩阵
            printf("\n重构矩阵:\n");
            for (int i = 0; i < 3; ++i) {
                printf("[%8.4f %8.4f %8.4f]\n", 
                        reconstructed[i*3], reconstructed[i*3+1], reconstructed[i*3+2]);
            }

            // print 差异
            printf("index: %d 总差异: %.6f\n", t, diff);
        }

    }

    if(IS_PRINT_ERROR){
        printf("总差异: %.6f\n", all_error);
        printf("average diff: %.6f\n", all_error / numTests);
    }

    // 清理内存
    free(h_matrices);
    free(h_eigenvalues);
    free(h_eigenvectors);
    hipFree(d_matrices);
    hipFree(d_eigenvalues);
    hipFree(d_eigenvectors);

    return {milliseconds, all_error / numTests};
}

// int main() {
//     int count = 10000;
    
//     for(int i = 0 ; i < 10000; i++){
//         auto runtime = testOnce();
//     }

//     return 0;
// }


int main() {
    int count = 1000;
    std::vector<double> runtimes;

    // 收集运行时间
    double percent = 0;
    double all_error = 0;
    for (int i = 0; i < count; i++) {
        auto [runtime, avg_error] = testOnce();
        runtimes.push_back(runtime);

        percent = (i + 1) * 100.0 / count;
        if (i % (int)(count/100) == 0) {
            std::cout << "进度: " << percent << "%\r" << std::flush;
        }
        all_error += avg_error;
    }


    // 排序
    std::sort(runtimes.begin(), runtimes.end());

    // 去掉最大值和最小值
    if (count > 2) {
        runtimes.erase(runtimes.begin());
        runtimes.pop_back();
    }

    // 计算统计量
    if (!runtimes.empty()) {
        double sum = 0.0;
        std::map<double, int> freq_map;

        // 计算总和和频率
        for (double rt : runtimes) {
            sum += rt;
            freq_map[rt]++;
        }

        double mean = sum / runtimes.size();

        // 计算方差
        double variance = 0.0;
        for (double rt : runtimes) {
            variance += std::pow(rt - mean, 2);
        }
        variance /= runtimes.size();

        // 找最大值和最小值
        double max_val = runtimes.back();
        double min_val = runtimes.front();

        // 找众数
        double mode = runtimes[0];
        int max_count = 1;
        for (const auto& pair : freq_map) {
            if (pair.second > max_count) {
                max_count = pair.second;
                mode = pair.first;
            }
        }

        // 输出结果
        std::cout << "样本数量（去掉最大最小值后）: " << runtimes.size() << std::endl;
        std::cout << "最大值: " << max_val << " 毫秒" << std::endl;
        std::cout << "最小值: " << min_val << " 毫秒" << std::endl;
        std::cout << "均值: " << mean << " 毫秒" << std::endl;
        std::cout << "众数(保留整数): " << static_cast<int>(mode) << " 毫秒" << std::endl;
        std::cout << "方差: " << variance << std::endl;
        std::cout << "标准差: " << std::sqrt(variance) << std::endl;

        std::cout << "平均差异: " << all_error / count << std::endl;
    } else {
        std::cout << "没有有效的运行时间数据。" << std::endl;
    }

    /*
    927000 x 3 x 3 dim
    样本数量（去掉最大最小值后）: 998
    最大值: 24.702 毫秒
    最小值: 13.5277 毫秒
    均值: 14.7503 毫秒
    众数(保留整数): 14 毫秒
    方差: 0.444104
    标准差: 0.666411
    平均差异: -2.14579e-06
    */

    return 0;
}



